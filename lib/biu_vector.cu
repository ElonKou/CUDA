#include "hip/hip_runtime.h"
/*================================================================
*  Copyright (C)2019 All rights reserved.
*  FileName : biu_vector.cu
*  Author   : ElonKou
*  Email    : elonkou@ktime.cc
*  Date     : 2019年08月17日 星期六 17时43分21秒
================================================================*/

#include "biu.hh"
#include "hip/hip_runtime.h"
#include ""

#define PREFIX __global__ void
#define VECTOR_ADD(TYPE)                                                       \
    __global__ void vector_add_##TYPE(TYPE *A, TYPE *B, TYPE *C, int N) {      \
        int idx = blockIdx.x * blockDim.x + threadIdx.x;                       \
        if (idx < N) {                                                         \
            C[idx] = A[idx] + B[idx];                                          \
        }                                                                      \
    }                                                                          \
    extern "C" void add_##TYPE(TYPE *A, TYPE *B, TYPE *C, int N) {             \
        int nBytes = N * sizeof(TYPE);                                         \
        int *cuda_A, *cuda_B, *cuda_C;                                         \
        hipMalloc((void **)&cuda_A, nBytes);                                  \
        hipMalloc((void **)&cuda_B, nBytes);                                  \
        hipMalloc((void **)&cuda_C, nBytes);                                  \
        hipMemcpy((void *)cuda_A, (void *)A, nBytes, hipMemcpyHostToDevice); \
        hipMemcpy((void *)cuda_B, (void *)B, nBytes, hipMemcpyHostToDevice); \
        dim3 blockSize(CUDA_W, CUDA_H);                                        \
        dim3 gridSize((N + CUDA_W - 1) / CUDA_W, (N + CUDA_H - 1) / CUDA_H);   \
        vector_add_int<<<gridSize, blockSize>>>(cuda_A, cuda_B, cuda_C, N);    \
        hipMemcpy((void *)C, (void *)cuda_C, nBytes, hipMemcpyDeviceToHost); \
        hipFree(cuda_A);                                                      \
        hipFree(cuda_B);                                                      \
        hipFree(cuda_C);                                                      \
    }

#define VECTOR_ADD_CONST(TYPE_A, TYPE_B)                                       \
    __global__ void vector_add_##TYPE_A##_##TYPE_B(TYPE_A *A, TYPE_B C,        \
                                                   int N) {                    \
        int idx = blockIdx.x * blockDim.x + threadIdx.x;                       \
        if (idx < N) {                                                         \
            A[idx] += C;                                                       \
        }                                                                      \
    }                                                                          \
    extern "C" void add_const_##TYPE_A##_##TYPE_B(TYPE_A *A, TYPE_B C,         \
                                                  int N) {                     \
        int nBytes = N * sizeof(TYPE_A);                                       \
        TYPE_A *cuda_A;                                                        \
        hipMalloc((void **)&cuda_A, nBytes);                                  \
        hipMemcpy((void *)cuda_A, (void *)A, nBytes, hipMemcpyHostToDevice); \
        dim3 blockSize(CUDA_W, CUDA_H);                                        \
        dim3 gridSize((N + CUDA_W - 1) / CUDA_W, (N + CUDA_H - 1) / CUDA_H);   \
        vector_add_##TYPE_A##_##TYPE_B<<<gridSize, blockSize>>>(cuda_A, C, N); \
        hipMemcpy((void *)A, (void *)cuda_A, nBytes, hipMemcpyDeviceToHost); \
        hipFree(cuda_A);                                                      \
    }

VECTOR_ADD(int)
VECTOR_ADD(float)
VECTOR_ADD(double)
VECTOR_ADD_CONST(int, int)
VECTOR_ADD_CONST(float, float)
VECTOR_ADD_CONST(double, double)