
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void add(int *a, int *b, int *c){
    if(threadIdx.x < N){
        c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
    }
}

int main(){
    int A[N], B[N], C[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **)&dev_a, N * sizeof(int));
    hipMalloc((void **)&dev_b, N * sizeof(int));
    hipMalloc((void **)&dev_c, N * sizeof(int));

    for(int i =0;i < N; i++){
        A[i] = -i;
        B[i] = i * i;
        C[i] = 0;
    }

    hipMemcpy(dev_a, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, B, N * sizeof(int), hipMemcpyHostToDevice);
    add<<<1, 10>>>(dev_a, dev_b, dev_c);
    hipMemcpy(C, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0;i < N; i++){
        printf("%d + %d = %d\n", A[i], B[i], C[i]);
    }
    return 0;
}