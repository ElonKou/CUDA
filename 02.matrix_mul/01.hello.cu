#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void SayHello(){
    printf("Hello from GPU:%d\n", threadIdx.x);
}

int main(){
    SayHello<<<1,10>>>();
    // cudaDeviceSynchronize();
    hipDeviceReset(); 
    return 0;
}

